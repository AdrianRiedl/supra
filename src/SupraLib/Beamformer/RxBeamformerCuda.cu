#include "hip/hip_runtime.h"
// ================================================================================================
// 
// If not explicitly stated: Copyright (C) 2016, all rights reserved,
//      Rüdiger Göbl 
//		Email r.goebl@tum.de
//      Chair for Computer Aided Medical Procedures
//      Technische Universität München
//      Boltzmannstr. 3, 85748 Garching b. München, Germany
// 
// ================================================================================================
#include "RxBeamformerCuda.h"
#include "USImage.h"
#include "USRawData.h"
#include "RxSampleBeamformerDelayAndSum.h"
#include "RxSampleBeamformerDelayMultiplyAndSum.h"
#include "RxSampleBeamformerDelayMultiplyAndSum2.h"
#include "RxSampleBeamformerDelayMultiplyAndSum3.h"
#include "RxSampleBeamformerDelayMultiplyAndSum4.h"
#include "RxSampleBeamformerSignedDelayMultiplyAndSum.h"
#include "RxSampleBeamformerDelayAndStdDev.h"
#include "RxSampleBeamformerTestSignal.h"
#include "RxBeamformerCommon.h"
#include "utilities/cudaUtility.h"
#include <utilities/Logging.h>

//TODO ALL ELEMENT/SCANLINE Y positons are actually Z! Change all variable names accordingly
namespace supra {
    using namespace logging;

    RxBeamformerCuda::RxBeamformerCuda(const RxBeamformerParameters &parameters)
            : m_windowFunction(nullptr) {
        m_lastSeenDt = 0;
        m_numRxScanlines = parameters.getNumRxScanlines();
        m_rxScanlineLayout = parameters.getRxScanlineLayout();

        m_is3D = (m_rxScanlineLayout.x > 1 && m_rxScanlineLayout.y > 1);
        m_speedOfSoundMMperS = parameters.getSpeedOfSoundMMperS();
        m_rxNumDepths = parameters.getRxNumDepths();

        // create and fill new buffers
        m_pRxDepths = std::unique_ptr < Container < LocationType > > (
                new Container<LocationType>(LocationGpu, hipStreamDefault, parameters.getRxDepths()));

        m_pRxScanlines = std::unique_ptr < Container < ScanlineRxParameters3D > > (
                new Container<ScanlineRxParameters3D>(LocationGpu, hipStreamDefault, parameters.getRxScanlines()));

        m_pRxElementXs = std::unique_ptr < Container < LocationType > > (
                new Container<LocationType>(LocationGpu, hipStreamDefault, parameters.getRxElementXs()));
        m_pRxElementYs = std::unique_ptr < Container < LocationType > > (
                new Container<LocationType>(LocationGpu, hipStreamDefault, parameters.getRxElementYs()));
    }

    RxBeamformerCuda::~RxBeamformerCuda() {
    }

    /*!
     * TODO what does this function do exactly?
     *
     * @param dt
     * @param speedOfSoundMMperS
     * @param numTransducerElements
     */
    void RxBeamformerCuda::convertToDtSpace(double dt, double speedOfSoundMMperS, size_t numTransducerElements) const {
        if (m_lastSeenDt != dt || m_speedOfSoundMMperS != speedOfSoundMMperS) {
            double oldFactor = 1;
            double oldFactorTime = 1;
            if (m_lastSeenDt != 0 && m_speedOfSoundMMperS != 0) {
                oldFactor = 1 / (m_speedOfSoundMMperS * m_lastSeenDt);
                oldFactorTime = 1 / m_lastSeenDt;
            }

            double factor = 1 / oldFactor / (speedOfSoundMMperS * dt);
            double factorTime = 1 / oldFactorTime / dt;

            m_pRxScanlines = std::unique_ptr < Container < ScanlineRxParameters3D > >
                             (new Container<ScanlineRxParameters3D>(LocationHost, *m_pRxScanlines));
            for (size_t i = 0; i < m_numRxScanlines; i++) {
                ScanlineRxParameters3D p = m_pRxScanlines->get()[i];
                p.position = p.position * factor;
                for (size_t k = 0; k < std::extent<decltype(p.txWeights)>::value; k++) {
                    p.txParameters[k].initialDelay *= factorTime;
                }
                p.maxElementDistance = p.maxElementDistance * factor;
                m_pRxScanlines->get()[i] = p;
            }
            m_pRxScanlines = std::unique_ptr < Container < ScanlineRxParameters3D > >
                             (new Container<ScanlineRxParameters3D>(LocationGpu, *m_pRxScanlines));

            m_pRxDepths = std::unique_ptr < Container < LocationType > >
                          (new Container<LocationType>(LocationHost, *m_pRxDepths));
            for (size_t i = 0; i < m_rxNumDepths; i++) {
                m_pRxDepths->get()[i] = static_cast<LocationType>(m_pRxDepths->get()[i] * factor);
            }
            m_pRxDepths = std::unique_ptr < Container < LocationType > >
                          (new Container<LocationType>(LocationGpu, *m_pRxDepths));

            m_pRxElementXs = std::unique_ptr < Container < LocationType > >
                             (new Container<LocationType>(LocationHost, *m_pRxElementXs));
            m_pRxElementYs = std::unique_ptr < Container < LocationType > >
                             (new Container<LocationType>(LocationHost, *m_pRxElementYs));
            for (size_t i = 0; i < numTransducerElements; i++) {
                m_pRxElementXs->get()[i] = static_cast<LocationType>(m_pRxElementXs->get()[i] * factor);
                m_pRxElementYs->get()[i] = static_cast<LocationType>(m_pRxElementYs->get()[i] * factor);
            }
            m_pRxElementXs = std::unique_ptr < Container < LocationType > >
                             (new Container<LocationType>(LocationGpu, *m_pRxElementXs));
            m_pRxElementYs = std::unique_ptr < Container < LocationType > >
                             (new Container<LocationType>(LocationGpu, *m_pRxElementYs));

            m_lastSeenDt = dt;
            m_speedOfSoundMMperS = speedOfSoundMMperS;
        }
    }

    /*!
     * TODO what does this function do exactly?
     * The result is written to the corresponding position in the memory which can be addressed by s.
     *
     * @tparam SampleBeamformer
     * @tparam interpolateRFlines               boolean about interpolated RF lines
     * @tparam interpolateBetweenTransmits      boolean about interpolation between the transmits
     * @tparam maxNumElements
     * @tparam maxNumFunctionElements
     * @tparam RFType
     * @tparam ResultType
     * @tparam LocationType
     * @param numTransducerElements
     * @param elementLayout
     * @param numReceivedChannels
     * @param numTimesteps
     * @param RF                                pointer to the raw data container
     * @param numTxScanlines                    number of transmitting scanlines
     * @param numRxScanlines                    number of receiving scanlines
     * @param scanlinesDT
     * @param numDs
     * @param dsDT
     * @param x_elemsDT
     * @param z_elemsDT
     * @param speedOfSound                      speed of the sound in the tissue
     * @param dt
     * @param additionalOffset
     * @param F
     * @param windowFunction
     * @param s                                 pointer to the allocated memory for the results
     */
    template<class SampleBeamformer, bool interpolateRFlines, bool interpolateBetweenTransmits, unsigned int maxNumElements, unsigned int maxNumFunctionElements, typename RFType, typename ResultType, typename LocationType>
    __global__
    void rxBeamformingDTSPACE3DKernel(
            uint32_t numTransducerElements,
            vec2T <uint32_t> elementLayout,
            uint32_t numReceivedChannels,
            uint32_t numTimesteps,
            const RFType *__restrict__ RF,
            uint32_t numTxScanlines,
            uint32_t numRxScanlines,
            const ScanlineRxParameters3D *__restrict__ scanlinesDT,
            uint32_t numDs,
            const LocationType *__restrict__ dsDT,
            const LocationType *__restrict__ x_elemsDT,
            const LocationType *__restrict__ z_elemsDT,
            LocationType speedOfSound,
            LocationType dt,
            uint32_t additionalOffset,
            LocationType F,
            const WindowFunctionGpu windowFunction,
            ResultType *__restrict__ s) {
        __shared__
        LocationType x_elemsDTsh[maxNumElements];
        __shared__
        LocationType z_elemsDTsh[maxNumElements];
        __shared__
        WindowFunction::ElementType functionShared[maxNumFunctionElements];
        //fetch element positions to shared memory
        for (int threadId = (threadIdx.y * blockDim.x) +
                            threadIdx.x;  //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
             threadId < maxNumElements && threadId < numTransducerElements;
             threadId += blockDim.x *
                         blockDim.y)  //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
        {
            x_elemsDTsh[threadId] = x_elemsDT[threadId];
            z_elemsDTsh[threadId] = z_elemsDT[threadId];
        }
        for (int threadId = (threadIdx.y * blockDim.x) +
                            threadIdx.x;  //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
             threadId < maxNumFunctionElements && threadId < windowFunction.numElements();
             threadId += blockDim.x *
                         blockDim.y)  //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
        {
            functionShared[threadId] = windowFunction.getDirect(threadId);
        }
        __syncthreads(); //@suppress("Function cannot be resolved")

        int r = blockDim.y * blockIdx.y +
                threadIdx.y; //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
        int scanlineIdx = blockDim.x * blockIdx.x +
                          threadIdx.x; //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")

        if (r < numDs && scanlineIdx < numRxScanlines) {
            LocationType d = dsDT[r];
            //TODO should this also depend on the angle?
            LocationType aDT = squ(computeAperture_D(F, d * dt * speedOfSound) / speedOfSound / dt);
            ScanlineRxParameters3D scanline = scanlinesDT[scanlineIdx];

            LocationType scanline_x = scanline.position.x;
            LocationType scanline_z = scanline.position.z;
            LocationType dirX = scanline.direction.x;
            LocationType dirY = scanline.direction.y;
            LocationType dirZ = scanline.direction.z;
            vec2f maxElementDistance = static_cast<vec2f>(scanline.maxElementDistance);
            vec2f invMaxElementDistance = vec2f{1.0f, 1.0f} / min(vec2f{sqrt(aDT), sqrt(aDT)}, maxElementDistance);

            float sInterp = 0.0f;

            int highestWeightIndex;
            if (!interpolateBetweenTransmits) {
                highestWeightIndex = 0;
                float highestWeight = scanline.txWeights[0];
                for (int k = 1; k < std::extent<decltype(scanline.txWeights)>::value; k++) {
                    if (scanline.txWeights[k] > highestWeight) {
                        highestWeight = scanline.txWeights[k];
                        highestWeightIndex = k;
                    }
                }
            }

            // now iterate over all four txScanlines to interpolate beamformed scanlines from those transmits
            for (int k = (interpolateBetweenTransmits ? 0 : highestWeightIndex);
                 (interpolateBetweenTransmits && k < std::extent<decltype(scanline.txWeights)>::value) ||
                 (!interpolateBetweenTransmits && k == highestWeightIndex);
                 k++) {
                if (scanline.txWeights[k] > 0.0) {
                    ScanlineRxParameters3D::TransmitParameters txParams = scanline.txParameters[k];
                    uint32_t txScanlineIdx = txParams.txScanlineIdx;
                    if (txScanlineIdx >= numTxScanlines) {
                        //ERROR!
                        return;
                    }
                    float sLocal = 0.0f;

                    sLocal = SampleBeamformer::template sampleBeamform3D<interpolateRFlines, RFType, float, LocationType>(
                            txParams, RF, elementLayout, numReceivedChannels, numTimesteps,
                            x_elemsDTsh, z_elemsDTsh, scanline_x, scanline_z, dirX, dirY, dirZ,
                            aDT, d, invMaxElementDistance, speedOfSound, dt, additionalOffset, &windowFunction,
                            functionShared);

                    if (interpolateBetweenTransmits) {
                        sInterp += static_cast<float>(scanline.txWeights[k]) * sLocal;
                    } else {
                        sInterp += sLocal;
                    }
                }
            }
            s[scanlineIdx + r * numRxScanlines] = clampCast<ResultType>(sInterp);
        }
    }
    /*!
     * TODO what does this function do exactly?
     * The result is written to the corresponding position in the memory which can be addressed by s.
     *
     * @tparam SampleBeamformer
     * @tparam interpolateRFlines               boolean about interpolated RF lines
     * @tparam interpolateBetweenTransmits      boolean about interpolation between the transmits
     * @tparam RFType
     * @tparam ResultType
     * @tparam LocationType
     * @param numTransducerElements
     * @param numReceivedChannels
     * @param numTimesteps
     * @param RF                                pointer to the raw data container
     * @param numTxScanlines                    number of transmitting scanlines
     * @param numRxScanlines                    number of receiving scanlines
     * @param scanlinesDT
     * @param numDs
     * @param dsDT
     * @param x_elemsDT
     * @param speedOfSound                      speed of the sound in the tissue
     * @param dt
     * @param additionalOffset
     * @param F
     * @param windowFunction
     * @param s                                 pointer to the allocated memory for the results
     */
    template<class SampleBeamformer, bool interpolateRFlines, bool interpolateBetweenTransmits, typename RFType, typename ResultType, typename LocationType>
    __global__
    void rxBeamformingDTSPACEKernel(
            size_t numTransducerElements,
            size_t numReceivedChannels,
            size_t numTimesteps,
            const RFType *__restrict__ RF,
            size_t numTxScanlines,
            size_t numRxScanlines,
            const ScanlineRxParameters3D *__restrict__ scanlinesDT,
            size_t numDs,
            const LocationType *__restrict__ dsDT,
            const LocationType *__restrict__ x_elemsDT,
            LocationType speedOfSound,
            LocationType dt,
            uint32_t additionalOffset,
            LocationType F,
            const WindowFunctionGpu windowFunction,
            ResultType *__restrict__ s) {
        int r = blockDim.y * blockIdx.y +
                threadIdx.y; //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
        int scanlineIdx = blockDim.x * blockIdx.x +
                          threadIdx.x; //@suppress("Symbol is not resolved") @suppress("Field cannot be resolved")
        if (r < numDs && scanlineIdx < numRxScanlines) {
            LocationType d = dsDT[r];
            //TODO should this also depend on the angle?
            LocationType aDT = computeAperture_D(F, d * dt * speedOfSound) / speedOfSound / dt;
            ScanlineRxParameters3D scanline = scanlinesDT[scanlineIdx];
            LocationType scanline_x = scanline.position.x;
            LocationType dirX = scanline.direction.x;
            LocationType dirY = scanline.direction.y;
            LocationType dirZ = scanline.direction.z;
            LocationType maxElementDistance = static_cast<LocationType>(scanline.maxElementDistance.x);
            LocationType invMaxElementDistance = 1 / min(aDT, maxElementDistance);

            float sInterp = 0.0f;

            int highestWeightIndex;
            if (!interpolateBetweenTransmits) {
                highestWeightIndex = 0;
                float highestWeight = scanline.txWeights[0];
                for (int k = 1; k < std::extent<decltype(scanline.txWeights)>::value; k++) {
                    if (scanline.txWeights[k] > highestWeight) {
                        highestWeight = scanline.txWeights[k];
                        highestWeightIndex = k;
                    }
                }
            }

            // now iterate over all four txScanlines to interpolate beamformed scanlines from those transmits
            for (int k = (interpolateBetweenTransmits ? 0 : highestWeightIndex);
                 (interpolateBetweenTransmits && k < std::extent<decltype(scanline.txWeights)>::value) ||
                 (!interpolateBetweenTransmits && k == highestWeightIndex);
                 k++) {
                if (scanline.txWeights[k] > 0.0) {
                    ScanlineRxParameters3D::TransmitParameters txParams = scanline.txParameters[k];
                    uint32_t txScanlineIdx = txParams.txScanlineIdx;
                    if (txScanlineIdx >= numTxScanlines) {
                        //ERROR!
                        return;
                    }

                    float sLocal = 0.0f;
                    sLocal = SampleBeamformer::template sampleBeamform2D<interpolateRFlines, RFType, float, LocationType>(
                            txParams, RF, numTransducerElements, numReceivedChannels, numTimesteps,
                            x_elemsDT, scanline_x, dirX, dirY, dirZ,
                            aDT, d, invMaxElementDistance, speedOfSound, dt, additionalOffset, &windowFunction);

                    if (interpolateBetweenTransmits) {
                        sInterp += static_cast<float>(scanline.txWeights[k]) * sLocal;
                    } else {
                        sInterp += sLocal;
                    }
                }
            }
            s[scanlineIdx + r * numRxScanlines] = clampCast<ResultType>(sInterp);
        }
    }
    /*!
     * The function calls the corresponding templated rxBeamformingDTSPACEKernel function.
     *
     * @tparam SampleBeamformer
     * @tparam maxWindowFunctionNumel
     * @tparam RFType
     * @tparam ResultType
     * @tparam LocationType
     * @param interpolateRFlines                boolean about interpolated RF lines
     * @param interpolateBetweenTransmits       boolean about interpolation between the transmits
     * @param numTransducerElements
     * @param elementLayout
     * @param numReceivedChannels
     * @param numTimesteps
     * @param RF                                pointer to the raw data container
     * @param numTxScanlines                    number of transmitting scanlines
     * @param numRxScanlines                    number of receiving scanlines
     * @param scanlines
     * @param numZs
     * @param zs
     * @param x_elems
     * @param y_elems
     * @param speedOfSound                      speed of the sound in the tissue
     * @param dt
     * @param additionalOffset
     * @param F
     * @param windowFunction
     * @param stream
     * @param s                                 pointer to the allocated memory for the results
     */
    template<class SampleBeamformer, unsigned int maxWindowFunctionNumel, typename RFType, typename ResultType, typename LocationType>
    void rxBeamformingDTspaceCuda3D(
            bool interpolateRFlines,
            bool interpolateBetweenTransmits,
            size_t numTransducerElements,
            vec2s elementLayout,
            size_t numReceivedChannels,
            size_t numTimesteps,
            const RFType *RF,
            size_t numTxScanlines,
            size_t numRxScanlines,
            const ScanlineRxParameters3D *scanlines,
            size_t numZs,
            const LocationType *zs,
            const LocationType *x_elems,
            const LocationType *y_elems,
            LocationType speedOfSound,
            LocationType dt,
            uint32_t additionalOffset,
            LocationType F,
            const WindowFunctionGpu windowFunction,
            hipStream_t stream,
            ResultType *s) {
        dim3 blockSize(1, 256);
        dim3 gridSize(
                static_cast<unsigned int>((numRxScanlines + blockSize.x - 1) / blockSize.x),
                static_cast<unsigned int>((numZs + blockSize.y - 1) / blockSize.y));

        if (interpolateRFlines) {
            if (interpolateBetweenTransmits) {
                logging::log_info("Setting rxBeamformingDTSPACE3DKernel true, true, 1024");
                rxBeamformingDTSPACE3DKernel<SampleBeamformer, true, true, 1024, maxWindowFunctionNumel> << <
                gridSize, blockSize, 0, stream >> > (
                        (uint32_t) numTransducerElements, static_cast<vec2T <uint32_t>>(elementLayout),
                                (uint32_t) numReceivedChannels, (uint32_t) numTimesteps, RF,
                                (uint32_t) numTxScanlines, (uint32_t) numRxScanlines, scanlines,
                                (uint32_t) numZs, zs, x_elems, y_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            } else {
                logging::log_info("Setting rxBeamformingDTSPACE3DKernel true, false, 1024");
                rxBeamformingDTSPACE3DKernel<SampleBeamformer, true, false, 1024, maxWindowFunctionNumel> << <
                gridSize, blockSize, 0, stream >> > (
                        (uint32_t) numTransducerElements, static_cast<vec2T <uint32_t>>(elementLayout),
                                (uint32_t) numReceivedChannels, (uint32_t) numTimesteps, RF,
                                (uint32_t) numTxScanlines, (uint32_t) numRxScanlines, scanlines,
                                (uint32_t) numZs, zs, x_elems, y_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            }
        } else {
            if (interpolateBetweenTransmits) {
                logging::log_info("Setting rxBeamformingDTSPACE3DKernel false, true, 1024");
                rxBeamformingDTSPACE3DKernel<SampleBeamformer, false, true, 1024, maxWindowFunctionNumel> << <
                gridSize, blockSize, 0, stream >> > (
                        (uint32_t) numTransducerElements, static_cast<vec2T <uint32_t>>(elementLayout),
                                (uint32_t) numReceivedChannels, (uint32_t) numTimesteps, RF,
                                (uint32_t) numTxScanlines, (uint32_t) numRxScanlines, scanlines,
                                (uint32_t) numZs, zs, x_elems, y_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            } else {
                logging::log_info("Setting rxBeamformingDTSPACE3DKernel false, false, 1024");
                rxBeamformingDTSPACE3DKernel<SampleBeamformer, false, false, 1024, maxWindowFunctionNumel> << <
                gridSize, blockSize, 0, stream >> > (
                        (uint32_t) numTransducerElements, static_cast<vec2T <uint32_t>>(elementLayout),
                                (uint32_t) numReceivedChannels, (uint32_t) numTimesteps, RF,
                                (uint32_t) numTxScanlines, (uint32_t) numRxScanlines, scanlines,
                                (uint32_t) numZs, zs, x_elems, y_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            }
        }
        cudaSafeCall(hipPeekAtLastError());
    }

    /*!
     * The function calls the corresponding templated rxBeamformingDTSPACEKernel function.
     *
     * @tparam SampleBeamformer
     * @tparam RFType
     * @tparam ResultType
     * @tparam LocationType
     * @param interpolateRFlines                boolean about interpolated RF lines
     * @param interpolateBetweenTransmits       boolean about interpolation between the transmits
     * @param numTransducerElements
     * @param numReceivedChannels
     * @param numTimesteps
     * @param RF                                pointer to the raw data container
     * @param numTxScanlines
     * @param numRxScanlines
     * @param scanlines
     * @param numZs
     * @param zs
     * @param x_elems
     * @param speedOfSound                      speed of the sound in the tissue
     * @param dt
     * @param additionalOffset
     * @param F
     * @param windowFunction
     * @param stream
     * @param s                                 pointer to the allocated memory for the results
     */
    template<class SampleBeamformer, typename RFType, typename ResultType, typename LocationType>
    void rxBeamformingDTspaceCuda(
            bool interpolateRFlines,
            bool interpolateBetweenTransmits,
            size_t numTransducerElements,
            size_t numReceivedChannels,
            size_t numTimesteps,
            const RFType *RF,
            size_t numTxScanlines,
            size_t numRxScanlines,
            const ScanlineRxParameters3D *scanlines,
            size_t numZs,
            const LocationType *zs,
            const LocationType *x_elems,
            LocationType speedOfSound,
            LocationType dt,
            uint32_t additionalOffset,
            LocationType F,
            const WindowFunctionGpu windowFunction,
            hipStream_t stream,
            ResultType *s) {
        dim3 blockSize(1, 256);
        dim3 gridSize(
                static_cast<unsigned int>((numRxScanlines + blockSize.x - 1) / blockSize.x),
                static_cast<unsigned int>((numZs + blockSize.y - 1) / blockSize.y));
        if (interpolateRFlines) {
            if (interpolateBetweenTransmits) {
                logging::log_info("Setting rxBeamformingDTSPACEKernel true, true");
                rxBeamformingDTSPACEKernel<SampleBeamformer, true, true> << < gridSize, blockSize, 0, stream >> > (
                        numTransducerElements, numReceivedChannels, numTimesteps, RF,
                                numTxScanlines, numRxScanlines, scanlines,
                                numZs, zs, x_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            } else {
                logging::log_info("Setting rxBeamformingDTSPACEKernel true, false");
                rxBeamformingDTSPACEKernel<SampleBeamformer, true, false> << < gridSize, blockSize, 0, stream >> > (
                        numTransducerElements, numReceivedChannels, numTimesteps, RF,
                                numTxScanlines, numRxScanlines, scanlines,
                                numZs, zs, x_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            }
        } else {
            if (interpolateBetweenTransmits) {
                logging::log_info("Setting rxBeamformingDTSPACEKernel false, true");
                rxBeamformingDTSPACEKernel<SampleBeamformer, false, true> << < gridSize, blockSize, 0, stream >> > (
                        numTransducerElements, numReceivedChannels, numTimesteps, RF,
                                numTxScanlines, numRxScanlines, scanlines,
                                numZs, zs, x_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            } else {
                logging::log_info("Setting rxBeamformingDTSPACEKernel false, false");
                rxBeamformingDTSPACEKernel<SampleBeamformer, false, false> << < gridSize, blockSize, 0, stream >> > (
                        numTransducerElements, numReceivedChannels, numTimesteps, RF,
                                numTxScanlines, numRxScanlines, scanlines,
                                numZs, zs, x_elems, speedOfSound, dt, additionalOffset, F, windowFunction, s);
            }
        }
        cudaSafeCall(hipPeekAtLastError());
    }


    /*!
     * The beamforming functions beamformingFunction3D and beamformingFunction2D are set according to the information in the GUI.
     * The according beamforming function is exectued with the necessary data.
     * The US image is created by using the values calculated by the beamforming function.
     *
     * @param sampleBeamformer                  defined in RxBeamformerCuda.h -> enumeration RxSampleBeamformer (DelayAndSum, DelayAndStdDev, DelayMultiplyAndSum, TestSignal)
     * @param rawData                           Pointer to the raw data
     * @param fNumber
     * @param speedOfSoundMMperS                speed of the sound in the tissue (mm/s)
     * @param windowType
     * @param windowParameter
     * @param interpolateBetweenTransmits       boolean about interpolation between the transmits
     * @param additionalOffset
     * @return                                  shared pointer to the USImage
     */
    template<typename ChannelDataType, typename ImageDataType>
    shared_ptr <USImage> RxBeamformerCuda::performRxBeamforming(
            RxBeamformerCuda::RxSampleBeamformer sampleBeamformer,
            shared_ptr<const USRawData> rawData,
            double fNumber,
            double speedOfSoundMMperS,
            WindowType windowType,
            WindowFunction::ElementType windowParameter,
            bool interpolateBetweenTransmits,
            int32_t additionalOffset) const {
        //Ensure the raw-data are on the gpu
        auto gRawData = rawData->getData<ChannelDataType>();
        if (!gRawData->isGPU() && !gRawData->isBoth()) {
            gRawData = std::make_shared < Container < ChannelDataType > > (LocationGpu, *gRawData);
        }

        size_t numelOut = m_numRxScanlines * m_rxNumDepths;
        shared_ptr <Container<ImageDataType>> pData = std::make_shared < Container < ImageDataType > >
                                                      (ContainerLocation::LocationGpu, gRawData->getStream(), numelOut);

        double dt = 1.0 / rawData->getSamplingFrequency();

        if (!m_windowFunction || m_windowFunction->getType() != windowType ||
            m_windowFunction->getParameter() != windowParameter) {
            m_windowFunction = std::unique_ptr<WindowFunction>(
                    new WindowFunction(windowType, windowParameter, m_windowFunctionNumEntries));
        }

        // Setting beamformingFunctions to Delay and Sum beamformer functions
        // For function refer to the RxSampleBeamformerDelayAndSum.h
        auto beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayAndSum, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
        auto beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayAndSum, ChannelDataType, ImageDataType, LocationType>;
        switch (sampleBeamformer) {
            case DelayAndSum:
                // Setting beamformingFunctions to Delay and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayAndSum.h
                logging::log_info("Setting Delay and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayAndSum, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayAndSum, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelayAndStdDev:
                // Setting beamformingFunctions to Delay and Standard Deviation beamformer functions
                // For function refers to the RxSampleBeamformerDelayAndStdDev.h
                logging::log_info("Setting Delay and StdDev beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayAndStdDev, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayAndStdDev, ChannelDataType, ImageDataType, LocationType>;
                break;
            case TestSignal:
                // Setting beamformingFunctions to Test Signal beamformer functions
                // For function refers to the RxSampleBeamformerTestSignal.h
                logging::log_info("Setting TestSignal beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerTestSignal, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerTestSignal, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelayMultiplyAndSum:
                // Setting beamformingFunctions to Delay Multiply and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayMultiplyAndSum.h
                logging::log_info("Setting Delay Multiply and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayMultiplyAndSum, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayMultiplyAndSum, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelayMultiplyAndSum2:
                // Setting beamformingFunctions to Delay Multiply and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayMultiplyAndSum.h
                logging::log_info("Setting Delay Multiply and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayMultiplyAndSum2, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayMultiplyAndSum2, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelayMultiplyAndSum3:
                // Setting beamformingFunctions to Delay Multiply and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayMultiplyAndSum.h
                logging::log_info("Setting Delay Multiply and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayMultiplyAndSum3, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayMultiplyAndSum3, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelayMultiplyAndSum4:
                // Setting beamformingFunctions to Delay Multiply and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayMultiplyAndSum.h
                logging::log_info("Setting Delay Multiply and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayMultiplyAndSum4, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayMultiplyAndSum4, ChannelDataType, ImageDataType, LocationType>;
                break;
            case DelaySignedMultiplyAndSum:
                // Setting beamformingFunctions to Delay Multiply and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayMultiplyAndSum.h
                logging::log_info("Setting Delay Multiply and Sum beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerSignedDelayMultiplyAndSum, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerSignedDelayMultiplyAndSum, ChannelDataType, ImageDataType, LocationType>;
                break;
            case INVALID:
            default:
                // Setting beamformingFunctions to Delay and Sum beamformer functions
                // For function refers to the RxSampleBeamformerDelayAndSum.h
                logging::log_info("Setting Invalid beamformingFunction");
                beamformingFunction3D = &rxBeamformingDTspaceCuda3D<RxSampleBeamformerDelayAndSum, m_windowFunctionNumEntries, ChannelDataType, ImageDataType, LocationType>;
                beamformingFunction2D = &rxBeamformingDTspaceCuda<RxSampleBeamformerDelayAndSum, ChannelDataType, ImageDataType, LocationType>;
        }


        convertToDtSpace(dt, speedOfSoundMMperS, rawData->getNumElements());
        if (m_is3D) {
            logging::log_info("Calling beamformingFunction 3D");
            beamformingFunction3D(
                    true,
                    interpolateBetweenTransmits,
                    rawData->getNumElements(),
                    rawData->getElementLayout(),
                    rawData->getNumReceivedChannels(),
                    rawData->getNumSamples(),
                    gRawData->get(),
                    rawData->getNumScanlines(), // numTxScanlines
                    m_numRxScanlines,            // numRxScanlines
                    m_pRxScanlines->get(),
                    m_rxNumDepths, m_pRxDepths->get(),
                    m_pRxElementXs->get(),
                    m_pRxElementYs->get(),
                    static_cast<LocationType>(m_speedOfSoundMMperS),
                    static_cast<LocationType>(dt),
                    additionalOffset,
                    static_cast<LocationType>(fNumber),
                    *(m_windowFunction->getGpu()),
                    gRawData->getStream(),
                    pData->get()
            );
        } else {
            logging::log_info("Calling beamformingFunction 2D");
            beamformingFunction2D(
                    true,
                    interpolateBetweenTransmits,
                    rawData->getNumElements(),
                    rawData->getNumReceivedChannels(),
                    rawData->getNumSamples(),
                    gRawData->get(),
                    rawData->getNumScanlines(), // numTxScanlines
                    m_numRxScanlines,            // numRxScanlines
                    m_pRxScanlines->get(),
                    m_rxNumDepths, m_pRxDepths->get(),
                    m_pRxElementXs->get(),
                    static_cast<LocationType>(m_speedOfSoundMMperS),
                    static_cast<LocationType>(dt),
                    additionalOffset,
                    static_cast<LocationType>(fNumber),
                    *(m_windowFunction->getGpu()),
                    gRawData->getStream(),
                    pData->get()
            );
        }

        if (rawData->getImageProperties() != m_lastSeenImageProperties) {
            m_lastSeenImageProperties = rawData->getImageProperties();
            shared_ptr <USImageProperties> newProps = std::make_shared<USImageProperties>(*m_lastSeenImageProperties);
            newProps->setScanlineLayout(m_rxScanlineLayout);
            newProps->setNumSamples(m_rxNumDepths);
            newProps->setImageState(USImageProperties::RF);
            m_editedImageProperties = std::const_pointer_cast<const USImageProperties>(newProps);
        }

        auto retImage = std::make_shared<USImage>(
                vec2s{m_numRxScanlines, m_rxNumDepths},
                pData,
                m_editedImageProperties,
                rawData->getReceiveTimestamp(),
                rawData->getSyncTimestamp());

        return retImage;
    }

    template
    shared_ptr <USImage> RxBeamformerCuda::performRxBeamforming<int16_t, int16_t>(
            RxBeamformerCuda::RxSampleBeamformer sampleBeamformer,
            shared_ptr<const USRawData> rawData,
            double fNumber,
            double speedOfSoundMMperS,
            WindowType windowType,
            WindowFunction::ElementType windowParameter,
            bool interpolateBetweenTransmits,
            int32_t additionalOffset) const;

    template
    shared_ptr <USImage> RxBeamformerCuda::performRxBeamforming<int16_t, float>(
            RxBeamformerCuda::RxSampleBeamformer sampleBeamformer,
            shared_ptr<const USRawData> rawData,
            double fNumber,
            double speedOfSoundMMperS,
            WindowType windowType,
            WindowFunction::ElementType windowParameter,
            bool interpolateBetweenTransmits,
            int32_t additionalOffset) const;

    template
    shared_ptr <USImage> RxBeamformerCuda::performRxBeamforming<float, int16_t>(
            RxBeamformerCuda::RxSampleBeamformer sampleBeamformer,
            shared_ptr<const USRawData> rawData,
            double fNumber,
            double speedOfSoundMMperS,
            WindowType windowType,
            WindowFunction::ElementType windowParameter,
            bool interpolateBetweenTransmits,
            int32_t additionalOffset) const;

    template
    shared_ptr <USImage> RxBeamformerCuda::performRxBeamforming<float, float>(
            RxBeamformerCuda::RxSampleBeamformer sampleBeamformer,
            shared_ptr<const USRawData> rawData,
            double fNumber,
            double speedOfSoundMMperS,
            WindowType windowType,
            WindowFunction::ElementType windowParameter,
            bool interpolateBetweenTransmits,
            int32_t additionalOffset) const;
}